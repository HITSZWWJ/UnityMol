#include <hip/hip_runtime.h>
//nvcc CudaAvailable.cu --shared -o CudaAvailability.dll

//
// Linearize OS specific macros
//
#if defined(__unix__) || defined(__linux__) || defined(__APPLE__) || defined(__MACH__)
#define OS_UNIX
#endif

#if defined(__APPLE__) || defined(__MACH__)
#define OS_OSX
#endif

#if defined(_MSC_VER) || defined(_WIN32) || defined(__CYGWIN__)
#define OS_WINDOWS
#endif

//
// API export macro
//
#if defined(OS_OSX)
#define API __attribute__((visibility("default")))
#elif defined(OS_WINDOWS)
#define API __declspec(dllexport)
#else
#define API
#endif

struct GpuCap
{
    bool QueryFailed;           // True on error
    int  DeviceCount;           // Number of CUDA devices found
    int  StrongestDeviceId;     // ID of best CUDA device
    int  ComputeCapabilityMajor; // Major compute capability (of best device)
    int  ComputeCapabilityMinor; // Minor compute capability
};





extern "C" {
    API bool isCudaAvailable() {
        GpuCap gpu;
        gpu.QueryFailed = false;
        gpu.StrongestDeviceId = -1;
        gpu.ComputeCapabilityMajor = -1;
        gpu.ComputeCapabilityMinor = -1;

        hipError_t error_id = hipGetDeviceCount(&gpu.DeviceCount);
        if (error_id != hipSuccess)
        {
            gpu.QueryFailed = true;
            gpu.DeviceCount = 0;
            return false;
        }
        if (gpu.DeviceCount == 0)
            return false;

        return true;
    }
}
